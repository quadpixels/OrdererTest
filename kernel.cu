#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <time.h>
#include <string.h>
#include <vector>
#include <device_atomic_functions.h>
#include <atomic>
#include <thread>
#include <mutex>
#include <assert.h>

std::mutex g_mutex;

struct Envelope
{
    char* Payload;
    char* Signature;
};

struct Batch
{
    Envelope* Messages;
    int MsgCount;
};

// Host <-> GPU mailbox
static std::thread* g_kernel_launcher_thread = nullptr;
struct MailboxEntry {
  Envelope* envelope;
  long len;
  int* condition_code;
  int index;
  int flag;
};

#define __HIPCC__

__device__ const long MaxPendingSizeBytes = 1048576;
__device__ const int MaxPendingMsgCount = 10;
__device__ const int MaxMsgSize = 10000; //Message can be this many chars, maybe need bigger?

//Pending batch structures hold messages being ordered until it's time to transfer them out
__device__ Batch PendingBatch1;
__device__ Batch PendingBatch2;
__device__ int WhichBatch = false; //False = PendingBatch1, true = PendingBatch2
__device__ long PendingBatchSizeBytes = 0;
__device__ int BatchFull = false;
__device__ int MsgNum = 0;

static MailboxEntry* mailbox_d; // Reference to mailbox_entries from the host side
std::atomic<int> g_mailbox_index = 0;

hipStream_t streams[100];

__global__ void Init()
{
    PendingBatch1.Messages = new Envelope[MaxPendingMsgCount];
    PendingBatch1.MsgCount = 0;
    PendingBatch2.Messages = new Envelope[MaxPendingMsgCount];
    PendingBatch2.MsgCount = 0;
    ////For testing
    //Envelope env1;
    //env1.Payload = "BQeUTS2rn9uC/XfzOus3aQ==";
    //env1.Signature = "/ULc6mTCl/oZlCJa4OkD0w==";
    //Envelope env2;
    //env2.Payload = "KMGirCuwYl8HGNuVjz3vPw==";
    //env2.Signature = "kyWJOTnbokNvmTOZiEX6mQ==";
    //PendingBatch1.Messages[0] = env1;
    //PendingBatch1.Messages[1] = env2;
    //PendingBatch1.MsgCount = 2;
}

__global__ void GetPendingCountGPU(int* pendingcnt)
{
    *pendingcnt = WhichBatch ? PendingBatch2.MsgCount : PendingBatch1.MsgCount;
}

extern "C" int __declspec(dllexport) __stdcall GetPendingCount()
{

    int* pendingcnt_d;
    hipMalloc(&pendingcnt_d, sizeof(int));
    int pendingcnt_h = 0;

    hipMemcpy(pendingcnt_d, &pendingcnt_h, sizeof(pendingcnt_h), hipMemcpyHostToDevice);

    GetPendingCountGPU<<<1, 1>>>(pendingcnt_d);

    hipMemcpy(&pendingcnt_h, pendingcnt_d, sizeof(pendingcnt_h), hipMemcpyDeviceToHost);

    return pendingcnt_h;

    ////Supposed to be able to use hipMemcpyFromSymbol to do this easier, but I kept getting error symbol unrecognized
    //int* cnt = new int;
    //hipError_t cudaStatus = hipMemcpyFromSymbol(cnt, HIP_SYMBOL(&PendingBatch.MsgCount), sizeof(int), 0, hipMemcpyHostToDevice);
    //if (cudaStatus != hipSuccess)
    //{
    //    int a = 1;
    //}
    //return *cnt;

}

__device__ char* strcpygpu(char* dest, const char* src) {
    int i = 0;
    do
    {
      dest[i] = src[i];
    } while (src[i++] != 0);
    return dest;
}

//Transfers byte strings from pending batch to output batch
__global__ void TransferStrings(Batch* b)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x; //Which message it is
    strcpygpu(b->Messages[i].Payload, WhichBatch ? PendingBatch1.Messages[i].Payload : PendingBatch2.Messages[i].Payload); //Which batch reversed since WhichBatch is toggled before running this
    strcpygpu(b->Messages[i].Signature, WhichBatch ? PendingBatch1.Messages[i].Signature : PendingBatch2.Messages[i].Signature); //Which batch reversed since WhichBatch is toggled before running this
}

//Transfers messages in pending batch to b, resets that pending batch, and 
__global__ void CutGPU(Batch* b)
{

    atomicExch(&WhichBatch, !WhichBatch); //Toggle which batch to use
    PendingBatchSizeBytes = 0; //Can't use atomicexch since type is long, maybe worth it to switch to int

    atomicExch(&BatchFull, false); //Now that WhichBatch has been toggled, can handle the previous batch while modifying the current one

                                   //Transfer pending batch to caller, reset pending batch
    TransferStrings<<<1, WhichBatch ? PendingBatch1.MsgCount : PendingBatch2.MsgCount>>>(b);

    b->MsgCount = WhichBatch ? PendingBatch1.MsgCount : PendingBatch2.MsgCount; //Which batch reversed since WhichBatch is toggled before running this

    atomicExch(WhichBatch ? &PendingBatch1.MsgCount : &PendingBatch2.MsgCount, 0); //Set msgcount to 0 for correct batch

}

//Retrieves the current pending batch
extern "C" Batch __declspec(dllexport) __stdcall Cut()
{
    //Batch allocation
    Batch* batch_d;
    hipMalloc(&batch_d, sizeof(*batch_d));
    Batch batch_h;
    //Envelope allocation
    Envelope* mess_d; //Array of size MaxPendingMsgCount
    hipMalloc(&mess_d, sizeof(*mess_d) * MaxPendingMsgCount);
    batch_h.Messages = mess_d;
    //Strings allocation
    std::vector<Envelope> msgs(MaxPendingMsgCount);
    for (auto& m : msgs)
    {
      hipMalloc(&m.Payload, MaxMsgSize);
      hipMalloc(&m.Signature, MaxMsgSize);
    }
    hipMemcpy(mess_d, msgs.data(), msgs.size() * sizeof(msgs[0]), hipMemcpyHostToDevice);
    //Copy to GPU
    hipMemcpy(batch_d, &batch_h, sizeof(*batch_d), hipMemcpyHostToDevice);
    //Run GPU function
    CutGPU<<<1, 1>>>(batch_d);
    //Batch back to CPU
    hipMemcpy(&batch_h, batch_d, sizeof(*batch_d), hipMemcpyDeviceToHost);
    //Messages back to CPU
    batch_h.Messages = new Envelope[MaxPendingMsgCount];
    hipMemcpy(batch_h.Messages, mess_d, sizeof(*mess_d) * MaxPendingMsgCount, hipMemcpyDeviceToHost);
    //Strings back to CPU
    for (int i = 0; i < batch_h.MsgCount; i++)
    {
      Envelope& eh = batch_h.Messages[i];
      eh = { (char*)malloc(MaxMsgSize), (char*)malloc(MaxMsgSize) };

      Envelope& ed = msgs[i]; // This has device pointers.

      hipMemcpy(eh.Payload, ed.Payload, MaxMsgSize, hipMemcpyDeviceToHost);
      hipMemcpy(eh.Signature, ed.Signature, MaxMsgSize, hipMemcpyDeviceToHost);
    }
    return batch_h;
    //*batch = batch_h;
}

//Adds msg to pending batch in GPU
__global__ void AddToPendingGPU(Envelope* msg, long len, int index)
{
    //  printf("Index: %i, MsgNum: %i, BatchFull: %d\n", index, MsgNum, BatchFull);

    //while (index > MsgNum || BatchFull) __nanosleep(100); //Wait until msg index equals next msg num and batch doesn't need to be cut
    while (index > MsgNum || BatchFull) {
      __threadfence();
    }

    if (WhichBatch)
    {
      PendingBatch2.Messages[atomicAdd(&PendingBatch2.MsgCount, 1)] = *msg; //Add message to pending batch, and increment msgcount
    }
    else
    {
      PendingBatch1.Messages[atomicAdd(&PendingBatch1.MsgCount, 1)] = *msg; //Add message to pending batch, and increment msgcount
    }
    PendingBatchSizeBytes += len;
    atomicAdd(&MsgNum, 1);
}

__device__ void AddToPendingGPU_device(Envelope* msg, long len, int index)
{
    printf("Index: %i, MsgNum: %i, BatchFull: %d\n", index, MsgNum, BatchFull);

    //while (index > MsgNum || BatchFull) __nanosleep(100); //Wait until msg index equals next msg num and batch doesn't need to be cut
    while (index > MsgNum || BatchFull) {
      __threadfence();
    }

    if (WhichBatch)
    {
        PendingBatch2.Messages[atomicAdd(&PendingBatch2.MsgCount, 1)] = *msg; //Add message to pending batch, and increment msgcount
    }
    else
    {
        PendingBatch1.Messages[atomicAdd(&PendingBatch1.MsgCount, 1)] = *msg; //Add message to pending batch, and increment msgcount
    }
    PendingBatchSizeBytes += len;
    atomicAdd(&MsgNum, 1);
}

//Adds msg to pending batch from CPU call
void AddToPending(Envelope msg, int index)
{
    //Memory for msg

    //Lengths and initial device and host vars
    long len = strlen(msg.Payload);
    long siglen = strlen(msg.Signature);
    Envelope* msg_d;
    hipMalloc(&msg_d, sizeof(*msg_d));
    Envelope msg_h;
    //Payload allocate and copy
    char* msgpayload_d;
    hipMalloc(&msgpayload_d, len);
    msg_h.Payload = msgpayload_d;
    hipMemcpy(msgpayload_d, msg.Payload, len, hipMemcpyHostToDevice);
    //Signature allocate and copy
    char* msgsig_d;
    hipMalloc(&msgsig_d, siglen);
    msg_h.Signature = msgsig_d;
    hipMemcpy(msgsig_d, msg.Signature, siglen, hipMemcpyHostToDevice);
    //Copy everything to device
    hipMemcpy(msg_d, &msg_h, sizeof(*msg_d), hipMemcpyHostToDevice);

    AddToPendingGPU<<<1, 1>>>(msg_d, len, index); //Call GPU function
}

//Condition denotes the situation caused by observing/adding the message and the current batch
//Condition is 0 by default, means do nothing
//If condition is 1 or 2, then conditions 3 or 4 can't be true
//If condition is 3, then condition 4 can't be true
__global__ void OrderedGPU(Envelope* msg, long len, int* ConditionCode, int index)
{
    printf("Hello world from the GPU.\n");
    //printf("Which batch: %i\n", WhichBatch);

    *ConditionCode = 0;

    //Message is too big and thus will overflow, send pending and this msg in its own batch
    if (len > MaxPendingSizeBytes)
    {
        //CONDITION 1, need to cut pending batch and isolate this msg
        if (WhichBatch ? PendingBatch2.MsgCount : PendingBatch1.MsgCount > 0)
        {
            *ConditionCode = 1;
        }
        //CONDITION 2, no need to cut pendign batch, still isolate this msg
        else
        {
            *ConditionCode = 2;
        }
        //printf("Hello World from GPU! %s\n", Batch2->Messages[0].Payload);
    }
    else
    {
        //CONDITION 3, Message will cause overflow, cut pending batch
        if (PendingBatchSizeBytes + len > MaxPendingSizeBytes)
        {
            *ConditionCode = 3;
            //Message still needs to be added, but it will be called manually from CPU
        }
        else //At this point condition code equals either 0 or 4
        {
            AddToPendingGPU_device(msg, len, index);

            //printf("PB1 cnt: %i\n", PendingBatch1.MsgCount);
            //printf("PB2 cnt: %i\n", PendingBatch2.MsgCount);

            //CONDITION 4 (only one that should occur during speed testing)
            //Pending batch has reached max count, must cut
            //If the last if statement was true, this one will not be true
            if((WhichBatch ? PendingBatch2.MsgCount : PendingBatch1.MsgCount) >= MaxPendingMsgCount)
            {
                //printf("Batch cut\n");
                atomicExch(&BatchFull, true);
                *ConditionCode = 4;
            }
        }
    }
}

__global__ void OrderedGPUUsingMailbox(MailboxEntry* mailbox_entries) {
  const int tid = blockIdx.x * blockDim.x + threadIdx.x; // Using only 1d thread grid
  if (tid == 0) {
    printf("[OrderedGPUUsingMailbox] %d blocks x %d threads per block\n",
      gridDim.x * gridDim.y * gridDim.z,
      blockDim.x * blockDim.y * blockDim.z);
  }

  volatile MailboxEntry* mailbox = &(mailbox_entries[tid]);
  volatile int* pflag = &(mailbox->flag);

//  printf("GPU thread #%d started, mailbox=%p, flag=%d\n", tid, mailbox, *pflag);

  bool done = false;
  while (!done) { // Don't use while(mailbox->flag == 0)
    if (*pflag != 0) done = true;
    //printf("GPU thread #%d, flag(%p)=%d\n", tid, pflag, *pflag);
  }

  printf("GPU thread #%d awakened by the mailbox\n", tid);

  Envelope* msg = mailbox->envelope;
  long len = mailbox->len;
  int* ConditionCode = mailbox->condition_code;
  int index = mailbox->index;

  // Same as below
  *ConditionCode = 0;

  //Message is too big and thus will overflow, send pending and this msg in its own batch
  if (len > MaxPendingSizeBytes)
  {
    //CONDITION 1, need to cut pending batch and isolate this msg
    if (WhichBatch ? PendingBatch2.MsgCount : PendingBatch1.MsgCount > 0)
    {
      *ConditionCode = 1;
    }
    //CONDITION 2, no need to cut pendign batch, still isolate this msg
    else
    {
      *ConditionCode = 2;
    }
    //printf("Hello World from GPU! %s\n", Batch2->Messages[0].Payload);
  }
  else
  {
    //CONDITION 3, Message will cause overflow, cut pending batch
    if (PendingBatchSizeBytes + len > MaxPendingSizeBytes)
    {
      *ConditionCode = 3;
      //Message still needs to be added, but it will be called manually from CPU
    }
    else //At this point condition code equals either 0 or 4
    {
      AddToPendingGPU_device(msg, len, index);

      //printf("PB1 cnt: %i\n", PendingBatch1.MsgCount);
      //printf("PB2 cnt: %i\n", PendingBatch2.MsgCount);

      //CONDITION 4 (only one that should occur during speed testing)
      //Pending batch has reached max count, must cut
      //If the last if statement was true, this one will not be true
      if((WhichBatch ? PendingBatch2.MsgCount : PendingBatch1.MsgCount) >= MaxPendingMsgCount)
      {
        //printf("Batch cut\n");
        atomicExch(&BatchFull, true);
        *ConditionCode = 4;
      }
    }
  }

  mailbox->flag = 2; // 2: completed
}


void CheckError(hipError_t err, const char* msg) {
  if (err != 0) {
    printf("%s: %s, %s\n", msg, hipGetErrorString(err), hipGetErrorName(err));
    abort();
  } else {
    //printf("No error: %s\n", msg);
  }
}

//Transfers msg into the OrderedGPU funtion which returns a condition code
//If msg is isolated in its own batch, Batch1 will be used. If a batch is cut, Batch2 will be used.
//Bool return value indicates if there is a pending batch. Batch1 and Batch2 are also written to and returned.
//Condition 0: Do nothing, msg just added to pending in ordered
//Condition 1: Must cut pending batch, and isolate msg in its own batch. The latter can easily be accomplished on the CPU.
//Condition 2: Don't cut pending batch, sitll isolate msg in its own batch. 
//Condition 3: Cut pending batch, then add msg. This means msg won't be added in OrderedGPU, and will have to have a separate kernel launch to add after cutting in this function.
//Condition 4: Cut pending batch after adding msg. This is the simplest one to handle.
//Also, there will be a pending batch after execution if condition is 1, 2, or 4.
extern "C" bool __declspec(dllexport) __stdcall Ordered(Envelope msg, Batch * Batch1, Batch * Batch2, int index)
{
    hipError_t err;
    printf("Ordered start: %i\n", index);

    //Memory for msg

    //Lengths and initial device and host vars
    long len = strlen(msg.Payload);
    long siglen = strlen(msg.Signature);
    Envelope* msg_d;
    err = hipMalloc(&msg_d, sizeof(*msg_d));
    CheckError(err, "1");
    Envelope msg_h;

    //Payload allocate and copy
    char* msgpayload_d;
    err = hipMalloc(&msgpayload_d, len);
    CheckError(err, "2");
    msg_h.Payload = msgpayload_d;
    err = hipMemcpy(msgpayload_d, msg.Payload, len, hipMemcpyHostToDevice);
    CheckError(err, "2");

    //Signature allocate and copy
    char* msgsig_d;
    err = hipMalloc(&msgsig_d, siglen);
    CheckError(err, "4");
    msg_h.Signature = msgsig_d;
    err = hipMemcpy(msgsig_d, msg.Signature, siglen, hipMemcpyHostToDevice);
    CheckError(err, "5");
    //Copy everything to device
    err = hipMemcpy(msg_d, &msg_h, sizeof(*msg_d), hipMemcpyHostToDevice);
    CheckError(err, "6");
    //Int for the condition code
    int* ConditionCode_d;
    err = hipMalloc(&ConditionCode_d, sizeof(int));
    CheckError(err, "7");
    int ConditionCode_h = 0;
    err = hipMemcpy(ConditionCode_d, &ConditionCode_h, sizeof(int), hipMemcpyHostToDevice);
    CheckError(err, "8");

    //////////Streams test. Currently has same result as using default stream

    //hipStreamCreate(&streams[index]);

    //hipStream_t test;

    //hipStreamCreateWithFlags(&test, hipStreamNonBlocking);

    ///******************Run GPU function******************/
    //OrderedGPU<<<1, 1, 0, test>>>(msg_d, len, ConditionCode_d, index);


    ////Copy condition code back to CPU
    //hipMemcpyAsync(&ConditionCode_h, ConditionCode_d, sizeof(int), hipMemcpyDeviceToHost, test);


    //////////Normal test

  /******************Run GPU function******************/
  if (true) {
    std::unique_lock<std::mutex> lock(g_mutex);
    if (g_kernel_launcher_thread == nullptr) {
      g_kernel_launcher_thread = new std::thread([](){
        hipError_t err;
        err = hipStreamCreateWithFlags(&(streams[1]), hipStreamNonBlocking);
        CheckError(err, "Creating Stream");
        err = hipStreamCreateWithFlags(&(streams[2]), hipStreamNonBlocking);
        CheckError(err, "Creating Stream 2");
        err = hipStreamCreateWithFlags(&(streams[3]), hipStreamNonBlocking);
        CheckError(err, "Creating Stream 3");
        //err = hipSetDeviceFlags(hipDeviceMapHost);
        //CheckError(err, "Set Device Flags");
        err = hipHostAlloc(&mailbox_d, sizeof(MailboxEntry)*100, hipHostMallocMapped);
        //err = hipHostAlloc(&mailbox_d, sizeof(MailboxEntry)*100, hipHostMallocMapped);
        CheckError(err, "Malloc mailbox entries");
        err = hipMemset(mailbox_d, 0, sizeof(MailboxEntry)*100);
        CheckError(err, "memset mailbox");
        void* mailbox_d1;
        //err = hipHostGetDevicePointer(&mailbox_d1, mailbox_d, 0);
        //CheckError(err, "Get device-side pointer of mailbox_d");
        //printf("%p vs %p\n", mailbox_d, mailbox_d1);
        
        printf("Mailbox is at %p\n", mailbox_d);
        OrderedGPUUsingMailbox<<<10, 1, 1, streams[2]>>>(mailbox_d);
        err = hipGetLastError();
        CheckError(err, "launch kernel");
        err = hipStreamSynchronize(streams[2]);
        CheckError(err, "stream sync");
      });
    }
  }

  if (0) {
    hipStreamCreate(&streams[index]);
    hipStream_t test;
    hipStreamCreateWithFlags(&test, hipStreamNonBlocking);
    OrderedGPU<<<1, 1>>>(msg_d, len, ConditionCode_d, index);
  } else {

    _sleep(200); // Wait until the setup is done

    int mailbox_index = g_mailbox_index.fetch_add(1);

    // Construct mailbox entry
    MailboxEntry me = { 0 };
    me.condition_code = ConditionCode_d;
    me.envelope = msg_d;
    me.len = len;
    me.index = index;
    me.flag = 1;
    
    err = hipMemcpyAsync(&(mailbox_d[mailbox_index]), &me, sizeof(me), hipMemcpyHostToDevice, streams[1]);
    CheckError(err, "Copying mailbox entry to the device");

    printf("Dispatching index=%d, mailbox_index=%d\n", index, mailbox_index);

    // Wait for the flag to be set to 2 by the GPU.
    while (true) {
      _sleep(100);
      me = mailbox_d[mailbox_index];
      if (me.flag == 2) break;
    }

    printf("Done index=%d mailbox_index=%d\n", index, mailbox_index);
  }

    //Copy condition code back to CPU
    hipMemcpy(&ConditionCode_h, ConditionCode_d, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(&msg_h);
    hipFree(ConditionCode_d);

    //Handle based on condition code
    //Doing it this way avoids unnecessarily copying batches in/out the GPU
    Batch1->MsgCount = 0; //Init to 0
    Batch2->MsgCount = 0;
    if (ConditionCode_h > 0) //Need to handle something
    {
        if (ConditionCode_h < 3) //Need to isolate msg in batch1
        {
            Batch1->Messages[0] = msg;
            Batch1->MsgCount = 1;
            if (ConditionCode_h == 1) //Also need to cut pending batch
            {
                *Batch2 = Cut();
            }
        }
        else //Condition code equals 3 or 4
        {
            *Batch2 = Cut();
            if (ConditionCode_h == 3) //Also need to add msg to pending after cutting
            {
                AddToPending(msg, index);
            }
        }
    }
    return ConditionCode_h == 0 || ConditionCode_h == 3;
}

extern "C" int __declspec(dllexport) __stdcall main()
{
    Init<<<1, 1>>>();

    ////Declaring randomly generated envelopes
    //Envelope env1;
    //env1.Payload = "BQeUTS2rn9uC/XfzOus3aQ==";
    //env1.Signature = "/ULc6mTCl/oZlCJa4OkD0w==";
    //Envelope env2;
    //env2.Payload = "KMGirCuwYl8HGNuVjz3vPw==";
    //env2.Signature = "kyWJOTnbokNvmTOZiEX6mQ==";
    //Envelope env3;
    //env3.Payload = "+qPLeEmLPkzffONM5wJr3A==";
    //env3.Signature = "pOPGu1/KmUweXz2RfcjXwQ==";

    /**********Ordered testing**********/
    //printf("Pending cnt: %i\n", GetPendingCount());

    /**********Ordered testing**********/
    //Envelope msg = env3;
    //Batch* b1 = new Batch();
    //Batch* b2 = new Batch();
    //clock_t start_t = clock();
    //bool ispending = Ordered(msg, b1, b2, 2);
    //printf("Hello: %i\n", b2->MsgCount);
    //clock_t end_t = clock();
    //double ticks = (double)(end_t - start_t);
    //printf("Time taken: %f\n", ticks);

    //////Cont.
    //msg = env1;
    //b1 = new Batch();
    //b2 = new Batch();
    //ispending = Ordered(msg, b1, b2, 3);
    //printf("Hello: %i\n", b2->MsgCount);

    //msg = env2;
    //b1 = new Batch();
    //b2 = new Batch();
    //ispending = Ordered(msg, b1, b2, 4);
    //printf("Hello: %i\n", b2->MsgCount);


    /**********Cut testing**********/
    //clock_t start_t = clock();
    //Batch test = Cut();
    //printf("Hello: %s\n", test.Messages[2].Payload);
    //clock_t end_t = clock();
    //double ticks = (double)(end_t - start_t);
    //printf("Time taken: %f\n", ticks);

    return 0;
}